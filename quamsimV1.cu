#include <stdio.h>
//#include <conio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <hip/hip_runtime.h>


#include <cmath>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

FILE *FP;    // to store trace file
char *input_file;
int i;

float num1,num2;
//float col1[2][2];
//float *col1[2];
float col2[4];
int index1,index2;
int qubit_circuit;
int qubit_oper;
int mask;
//float (*mat1)[2];
float (*mat2)[1];
float (*res)[1];
struct timeval begin, end;

void mat_mul(float **col1,float **vector_array, int size,int qubit_oper)
{
    //printf("why");

    int arr_bool[size];
	for(int i=0;i<size;i++){
		arr_bool[i]=0;
	}

	for(int j=0;j<size-1;j++)
	{
		float res_array[2][1];
		memset(res_array, 0, sizeof(res_array));
		if (arr_bool[j]==1){
			//printf("%d",i);
		}
		else
		{
			//float **vec_mat;
			float col2[2][1];
			mask = 1<<qubit_oper;
			index1 = j;
			index2 = mask ^j;
			col2[0][0]=vector_array[index1][0];
			col2[1][0]=vector_array[index2][0];
			mat2=col2;
			arr_bool[index1]=1;
			arr_bool[index2]=1;
            for (int i = 0; i < 2; ++i) {
                for (int j = 0; j < 1; ++j) {
                    for (int k = 0; k < 2; ++k) {
                    res_array[i][j] += col1[i][k] * mat2[k][j];
                }
      }
  }
  res=res_array;
    vector_array[index1][0]=res[0][0];
    vector_array[index2][0]=res[1][0];

		}
	}

	for (int i = 0; i < size; ++i) {
      for (int j = 0; j < 1; ++j) {
         printf("%.3f  ", vector_array[i][j]);
         printf("\n");
      }
  }
}
void mat_mul1(float* u,float* ip,float *op,int size,int qubit_oper){
    printf("%d\n",qubit_oper);
	
    for(int j=0;j<size-1;j++){
       
        mask = 1<<qubit_oper;
	    index1 = j;
		index2 = mask ^j;
		
		if(((j>>qubit_oper)&1)==0){
		op[index1]=(u[0]*ip[index1])+(u[1]*ip[index2]);
		op[index2]=(u[2]*ip[index1])+(u[3]*ip[index2]);}
		
    }
    for(int j=0;j<size-1;j++){printf("%.3f\n",op[j]);    }
    
}

__global__ void mat_mul(float *d_u, float *d_ip,float *d_op,int qubit)
{
		//for(int j=0;j<128;j++){printf("%f\n",d_op[j]);}
		int index1,index2;
		int mask;
		int i= blockDim.x * blockIdx.x + threadIdx.x;
		mask = 1<<qubit;
	    index1 = i;
		index2 = mask ^i;
		if(((i >>  qubit) & 1) == 0)
		{
			
			__shared__ float s1[2];
			__shared__ float s2[2];
			
			for(int j=0;j<3;j=j+2){
				s1[threadIdx.x]=d_u[j]*d_ip[i];
			}
		__syncthreads();
			for(int k=1;k<4;k=k+2)
			{
				s2[threadIdx.x]=d_u[k]*d_ip[i+(1<< qubit)];
			}
			__syncthreads();
			for(int q=0;q<1;q++)
			{
			d_op[i]=s1[q]+s2[q];
			d_op[i+(1<< qubit)] =s1[q+1]+s2[q+1];	
			}
			
			
			//d_op[i] = (d_u[0] * d_ip[i]) + (d_u[1] * d_ip[i+(1<< qubit)]);
			//d_op[i+(1<< qubit)] = (d_u[2] * d_ip[i]) + (d_u[3] * d_ip[i+(1<< qubit)]);
			//printf("%f\n",d_ip[i]);
			//printf("%f\n",d_ip[i+(1<<0)]);
			

		}
}



int main(int argc, char *argv[])
{
    //qubit_circuit = argv[0]; //qubit circuit
	//printf("%d\n",qubit_circuit);
    //scanf("%d",&qubit_circuit);
    //scanf("%d",&qubit_oper);
    //qubit_oper      = argv[1]; //qubit operation
    input_file = argv[1]; // "input_for_qc7_q0_q2_q3_q4_q5_q6.txt";
    //input_1=argv[0];
    FP = fopen(input_file, "r");
    if (FP == NULL)
    {
        printf("Unable to open file %s\n", input_file);
        return 1;
    }
	int coun=0;
    int p=0;



	int mask;
	//int size = pow(2,qubit_circuit);

    int count=0;
    //int p=0;
	while(fscanf(FP, "%f", &num1) != EOF){
    if(p>23)
        {
        count++;}
    p++;
	}
	//printf("%d,",p);
	//printf("%d",count);
	float **vector_array;
	float *ip;
	float *op;
	
	float *d_u,*d_ip,*d_op;
	int d_qopr;
	
	hipMalloc((void**)&d_u,4*sizeof(float));
    	hipMalloc((void**)&d_ip,(count-6)*sizeof(float));
    	hipMalloc((void**)&d_op,(count-6)*sizeof(float));
	
	int block_size = 256;
	 int grid_size = int(count/block_size);
	//dim3 grid(grid_size,grid_size);
	//dim3 threads(block_size, block_size);
	
	
	vector_array=(float**) malloc(sizeof(float*)*count-1);
	ip= (float*) malloc(sizeof(float)*count-6);
	op=(float*) malloc(sizeof(float)*count-6);
	float **col1;
	float *u1,*u2,*u3,*u4,*u5,*u6;
	int *qubit;
	col1=(float**)malloc(sizeof(float*)*2);
	u1=(float*)malloc(sizeof(float)*4);
	u2=(float*)malloc(sizeof(float)*4);
	u3=(float*)malloc(sizeof(float)*4);
	u4=(float*)malloc(sizeof(float)*4);
	u5=(float*)malloc(sizeof(float)*4);
	u6=(float*)malloc(sizeof(float)*4);
	qubit=(int*)malloc(sizeof(int)*6);
	int a=0;
	int b=0;
	int c=0;
	int d=0;
	int e=0;
	int f=0;
	int g=0;
	
	for(i=0;i<2;i++){
        col1[i]=(float*) malloc(sizeof(float)*2);
	}
    //printf("%d,",size);
    int test_size=count-6;
	for(i=0;i<count-1;i++){
        vector_array[i]=(float*) malloc(sizeof(float)*1);
        
	}
	op=(float*)malloc(sizeof(float)*test_size);
	ip=(float*)malloc(sizeof(float)*test_size);
	fseek(FP,0,SEEK_SET);
    //open trace file to read
	i=0;
	int l=0;


	while(fscanf(FP, "%f", &num1) != EOF)
    {
		if(i<4){
		col2[i]=num1;
		u1[i]=num1;
		//printf("%f",col2[i]);
		//printf("\n");
		}
		if(i>3 && i<8){
		u2[a]=num1;
		a++;
		}
		if(i>7 && i<12){
		u3[b]=num1;
		b++;
		}
		if(i>11 && i<16){
		u4[c]=num1;
		c++;
		}
		if(i>15 && i<20){
		u5[d]=num1;
		d++;
		}
		if(i>19 && i<24){
		u6[e]=num1;
		e++;
		}
		//printf("%f",col1[i][c]);
		//printf("\n");
		for(int a=0;a<2;a++){
		    for(int b=0;b<2;b++){
		        col1[a][b]=col2[(a*2)+b];		   }
		}
		

        

		if(i>23 && i<(p-6)){

		    vector_array[l][0]=num1;
		    ip[l]=num1;
		    l++;
		    
		}

        i++;
        if(i>p-6){
            qubit_oper =  num1;
		qubit[g]=num1;
		g++;
            

		}
	}
	
	int num_frag = (count-6)/32;
	float *inp;
	float *onp;
	int k=0;
	int n=0;
	dim3 grid(2,256);
	
	for(i=0;i<num_frag;i++)
	{
		for(int j=0;j<32;j++)
		{
			inp[j]=ip[k];
			k++;
		}
	hipMemcpy(d_u,u1,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,inp,32*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_op,onp,32*sizeof(float),hipMemcpyHostToDevice);
		
		mat_mul<<<grid, 32>>>(d_u,d_ip,d_op,qubit[0]);
	hipMemcpy(onp,d_op,32*sizeof(float),hipMemcpyDeviceToHost);
		for(int h=0;h<32;h++)
		{
			op[n]=onp[h]; 
			n++;
		}
	}
	
	for(i=0;i<count-6;i++)
	{
		ip[i]=op[i];
	}
	
	
	/*dim3 grid(2,256);
	
	 cudaMemcpy(d_u,u1,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	 
	 
	gettimeofday (&begin, NULL);
          
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[0]);
    gettimeofday (&end, NULL);
	
	cudaMemcpy(ip,d_op,(count-6)*sizeof(float),cudaMemcpyDeviceToHost);
	//for(int j=0;j<count-6;j++){printf("%.3f\n",ip[j]);    }
	cudaFree(d_u);
	cudaFree(d_op);
	cudaFree(d_ip);
	
	cudaMemcpy(d_u,u2,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[1]);
	
	cudaMemcpy(ip,d_op,(count-6)*sizeof(float),cudaMemcpyDeviceToHost);
	
	cudaMemcpy(d_u,u3,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[2]);
	
	cudaMemcpy(ip,d_op,(count-6)*sizeof(float),cudaMemcpyDeviceToHost);
	
	cudaMemcpy(d_u,u4,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[3]);
	
	cudaMemcpy(ip,d_op,(count-6)*sizeof(float),cudaMemcpyDeviceToHost);
	
	cudaMemcpy(d_u,u5,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[4]);
	
	cudaMemcpy(ip,d_op,(count-6)*sizeof(float),cudaMemcpyDeviceToHost);
	
	
	cudaMemcpy(d_u,u6,4*sizeof(float),cudaMemcpyHostToDevice);
	 cudaMemcpy(d_ip,ip,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(d_op,op,(count-6)*sizeof(float),cudaMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[5]);
	
	cudaMemcpy(op,d_op,(count-1)*sizeof(float),cudaMemcpyDeviceToHost);*/
	
	//mat_mul1(u,ip,op,count-1,qubit_oper);
	for(int j=0;j<count-6;j++){printf("%.3f\n",op[j]);    }
    fclose(FP);
}

