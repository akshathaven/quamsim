#include <stdio.h>
//#include <conio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <hip/hip_runtime.h>


#include <cmath>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

FILE *FP;    // to store trace file
char *input_file;
int i;

float num1,num2;
//float col1[2][2];
//float *col1[2];
float col2[4];
int index1,index2;
int qubit_circuit;
int qubit_oper;
int mask;
//float (*mat1)[2];
float (*mat2)[1];
float (*res)[1];
struct timeval begin, end;


__global__ void mat_mul(float *d_u, float *d_ip,float *d_op,int qubit)
{
		//for(int j=0;j<128;j++){printf("%f\n",d_op[j]);}
		int index1,index2;
		int mask;
		int i= blockDim.x * blockIdx.x + threadIdx.x;
		mask = 1<<qubit;
	    index1 = i;
		index2 = mask ^i;
		if(((i >>  qubit) & 1) == 0)
		{
			d_op[i] = (d_u[0] * d_ip[i]) + (d_u[1] * d_ip[i+(1<< qubit)]);
			d_op[i+(1<< qubit)] = (d_u[2] * d_ip[i]) + (d_u[3] * d_ip[i+(1<< qubit)]);
		//	printf("%f\n",d_ip[i]);
			//printf("%f\n",d_ip[i+(1<<0)]);
			

		}
}



int main(int argc, char *argv[])
{
    //qubit_circuit = argv[0]; //qubit circuit
	//printf("%d\n",qubit_circuit);
    //scanf("%d",&qubit_circuit);
    //scanf("%d",&qubit_oper);
    //qubit_oper      = argv[1]; //qubit operation
    input_file = argv[1]; // "input_for_qc7_q0_q2_q3_q4_q5_q6.txt";
    //input_1=argv[0];
    FP = fopen(input_file, "r");
    if (FP == NULL)
    {
        printf("Unable to open file %s\n", input_file);
        return 1;
    }
	int coun=0;
    int p=0;



	int mask;
	//int size = pow(2,qubit_circuit);

    int count=0;
    //int p=0;
	while(fscanf(FP, "%f", &num1) != EOF){
    if(p>23)
        {
        count++;}
    p++;
	}
	//printf("%d,",p);
	//printf("%d",count);
	float **vector_array;
	float *ip;
	float *op;
	
	float *d_u,*d_ip,*d_op;
	int d_qopr;
	
	hipMalloc((void**)&d_u,4*sizeof(float));
    	hipMalloc((void**)&d_ip,(count-6)*sizeof(float));
    	hipMalloc((void**)&d_op,(count-6)*sizeof(float));
	
	int block_size = 256;
	 int grid = (count-6)/64;
	//dim3 grid(grid_size,grid_size);
	//dim3 threads(block_size, block_size);
	
	
	vector_array=(float**) malloc(sizeof(float*)*count-1);
	ip= (float*) malloc(sizeof(float)*count-6);
	op=(float*) malloc(sizeof(float)*count-6);
	float **col1;
	float *u1,*u2,*u3,*u4,*u5,*u6;
	int *qubit;
	col1=(float**)malloc(sizeof(float*)*2);
	u1=(float*)malloc(sizeof(float)*4);
	u2=(float*)malloc(sizeof(float)*4);
	u3=(float*)malloc(sizeof(float)*4);
	u4=(float*)malloc(sizeof(float)*4);
	u5=(float*)malloc(sizeof(float)*4);
	u6=(float*)malloc(sizeof(float)*4);
	qubit=(int*)malloc(sizeof(int)*6);
	int a=0;
	int b=0;
	int c=0;
	int d=0;
	int e=0;
	int f=0;
	int g=0;
	
	for(i=0;i<2;i++){
        col1[i]=(float*) malloc(sizeof(float)*2);
	}
    //printf("%d,",size);
    int test_size=count-6;
	for(i=0;i<count-1;i++){
        vector_array[i]=(float*) malloc(sizeof(float)*1);
        
	}
	op=(float*)malloc(sizeof(float)*test_size);
	ip=(float*)malloc(sizeof(float)*test_size);
	fseek(FP,0,SEEK_SET);
    //open trace file to read
	i=0;
	int l=0;


	while(fscanf(FP, "%f", &num1) != EOF)
    {
		if(i<4){
		col2[i]=num1;
		u1[i]=num1;
		//printf("%f",col2[i]);
		//printf("\n");
		}
		if(i>3 && i<8){
		u2[a]=num1;
		a++;
		}
		if(i>7 && i<12){
		u3[b]=num1;
		b++;
		}
		if(i>11 && i<16){
		u4[c]=num1;
		c++;
		}
		if(i>15 && i<20){
		u5[d]=num1;
		d++;
		}
		if(i>19 && i<24){
		u6[e]=num1;
		e++;
		}
		//printf("%f",col1[i][c]);
		//printf("\n");
		for(int a=0;a<2;a++){
		    for(int b=0;b<2;b++){
		        col1[a][b]=col2[(a*2)+b];		   }
		}
		

        

		if(i>23 && i<(p-6)){

		    vector_array[l][0]=num1;
		    ip[l]=num1;
		    l++;
		    
		}

        i++;
        if(i>p-6){
            qubit_oper =  num1;
		qubit[g]=num1;
		g++;
            

		}
	}
	
	
	//dim3 grid(2,256);
	
	 hipMemcpy(d_u,u1,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,ip,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	 
	 
	gettimeofday (&begin, NULL);
          
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[0]);
    gettimeofday (&end, NULL);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);
	//for(int j=0;j<count-6;j++){printf("%.3f\n",op[j]);    }

	
	hipMemcpy(d_u,u2,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[1]);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);
 	
	
	hipMemcpy(d_u,u3,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[2]);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);
 
	
	hipMemcpy(d_u,u4,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[3]);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);

	
	hipMemcpy(d_u,u5,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[4]);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);
  
	
	
	hipMemcpy(d_u,u6,4*sizeof(float),hipMemcpyHostToDevice);
	 hipMemcpy(d_ip,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_op,op,(count-6)*sizeof(float),hipMemcpyHostToDevice);
	
	mat_mul<<<grid, 256>>>(d_u,d_ip,d_op,qubit[5]);
	
	hipMemcpy(op,d_op,(count-6)*sizeof(float),hipMemcpyDeviceToHost);
 for(int j=0;j<count-6;j++){printf("%.3f\n",op[j]);    }
	
 	//for(int j=0;j<count-6;j++){printf("%.3f\n",ip[j]);    }
	//mat_mul1(u,ip,op,count-1,qubit_oper);
	//for(int j=0;j<count-6;j++){printf("%.3f\n",op[j]);    }
 	hipFree(d_u);
	hipFree(d_op);
	hipFree(d_ip);
 //printf("WHhjkflg");
    fclose(FP);
}

